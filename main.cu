#include "hip/hip_runtime.h"
#include "grad/grad.h"
#include <chrono>

using timer = std::chrono::system_clock;

int main()
{
    /*auto ilist = { 1.f, 3.f, 3.f, 7.f, 2.f, 2.f, 8.f, 6.f, 9.f, 4.f, 2.f, 0.f, 1.f, 4.f, 8.f, 8.f };

    auto x = grad::make_array(grad::make_shape(ilist.size()), ilist);
    auto y = x.reshape(grad::make_shape(4, 4));
    auto w = y.reshape(grad::make_shape(4, 2, 2));
    auto u = w.reshape(grad::make_shape(2, 2, 2, 2));
    auto v = u.reshape(grad::make_shape(1, 16));

    std::cout << x << y << w << u << v;*/

    const size_t k = 11;
    size_t n[k] = { 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096, 8192 };

    //uint64_t time_alloc[k] = { 0 };
    uint64_t time_pure[k]  = { 0 };

    size_t s = 0;

    size_t tests = 1000;

    auto global_then = timer::now();
    for (int t = 0; t < tests; t++)
        for (int i = 0; i < k; i++)
        {
            auto m = n[i];
            auto x = grad::make_arithm(grad::make_shape(m, m), 0.f, 1.f);

            //auto then_alloc = timer::now();
            auto a = 3._fC / grad::asin(x);
            auto b = grad::pow(2._fC, grad::log(x));

            auto then_pure = timer::now();
            auto c = grad::matmul(a, b);

            s += c.size();

            auto now = timer::now();
            //time_alloc[i] += std::chrono::duration_cast<std::chrono::microseconds>(now - then_alloc).count();
            time_pure[i]  += std::chrono::duration_cast<std::chrono::microseconds>(now - then_pure).count();
        }

    std::cout << "tests = " << tests << ", k = " << k << "\n\n"
              << "full tests time = "
              << std::chrono::duration_cast<std::chrono::seconds>(timer::now() - global_then).count()
              << " s\n\n";
    for (int i = 0; i < k; i++)
        std::cout << "n = " << n[i] << '\n'
                  //<< "\tavg time with alloc  = " << time_alloc[i] / tests << " us\n"
                  << "\tavg time matmul = " << time_pure[i]  / tests << " us\n";

    std::cout << "\n" << s;

    std::cin.get();
}
