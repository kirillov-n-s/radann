#include "hip/hip_runtime.h"
#include "grad/array.h"
#include "grad/unary_ops.h"
#include "grad/binary_ops.h"
#include "grad/generators.h"

int main()
{
    grad::array<float, 2> x { grad::make_shape(4, 3), { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } };
    grad::array<float, 1> y { grad::make_shape(4), { -9, -9, -9, -9 } };
    grad::array<float, 0> z { grad::make_shape(), { 3.14 } };
    grad::array<float, 2> w = (x + grad::abs(y)) * -grad::cos(z);

    std::cout << x << y << z << w;

    std::cin.get();
}
